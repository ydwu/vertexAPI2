#include "hip/hip_runtime.h"
/******************************************************************************
Copyright 2013 Royal Caliber LLC. (http://www.royal-caliber.com)

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
******************************************************************************/


#include "refgas.h"
#include "gpugas.h"
#include "util.cuh"
#include "graphio.h"
#include <vector>
#include <iostream>


//Vertex program for Pagerank
struct PageRank
{
  static const float pageConst = 0.15f;
  static const float tol = 0.01f;

  struct VertexData
  {
    float rank;
    int   numOutEdges;
    friend std::ostream& operator<<(std::ostream &out, const VertexData &data);
  };

  struct EdgeData {};

  typedef float GatherResult;

  static const float gatherZero = 0.0f;

  __host__ __device__
  static float gatherMap(
    const VertexData* dst, const VertexData* src, const EdgeData* edge)
  {
    //this division is being done too many times right?
    //should just store the normalized value in apply?
    return src->rank / src->numOutEdges;
  }

  __host__ __device__
  static float gatherReduce(const float& left, const float& right)
  {
    return left + right;
  }

  __host__ __device__
  static bool apply(VertexData* vertexData, const float& gatherResult)
  {
    float newRank = pageConst + (1.0f - pageConst) * gatherResult;
    bool ret = fabs(newRank - vertexData->rank) >= tol;
    vertexData->rank = newRank;
    return ret;
  }

  __host__ __device__
  static void scatter(
    const VertexData* src, const VertexData *dst, EdgeData* edge)
  {
    //nothing
  }
};


void outputRanks(
  int n, const PageRank::VertexData* vertexData, FILE* f = stdout)
{
  for( int i = 0; i < n; ++i )
  {
    fprintf(f, "%d %f\n", i, vertexData[i].rank);
  }
}


template<typename Engine>
void run(int nVertices, PageRank::VertexData* vertexData, int nEdges
  , const int* srcs, const int* dsts)
{
  Engine engine;

  GpuTimer gpu_timer;
  float elapsed = 0.0f;
  int iteration = 0;

  // average elapsed time of 10 runs
  int num_runs = 10;
  for (int itr = 0; itr < num_runs; ++itr)
  {
    // reset graph
    for( int i = 0; i < nVertices; ++i )
      vertexData[i].rank = PageRank::pageConst;

    engine.setGraph(nVertices, vertexData, nEdges, 0, srcs, dsts);
    //all vertices begin active for pagerank
    engine.setActive(0, nVertices);

    gpu_timer.Start();

    while (engine.countActive())
    {
      engine.gatherApply();
      engine.scatterActivate();
      engine.nextIter();
      ++iteration;
    }

    engine.getResults();

    gpu_timer.Stop();
    elapsed += gpu_timer.ElapsedMillis();
  }

  elapsed /= num_runs;
  printf("num_iteration: %d\n", iteration);
  printf("elapsed: %f ms\n", elapsed);
}


int main(int argc, char **argv)
{
  char* inputFilename;
  char* outputFilename = 0;
  bool runTest;
  bool dumpResults;
  if( !parseCmdLineSimple(argc, argv, "s-t-d|s"
    , &inputFilename, &runTest, &dumpResults, &outputFilename) )
  {
    printf("Usage: pagerank [-t] [-d] inputfile [outputfile]\n");
    exit(1);
  }

  //load the graph
  int nVertices;
  std::vector<int> srcs;
  std::vector<int> dsts;
  loadGraph(inputFilename, nVertices, srcs, dsts);
  printf("loaded %s with %d vertices and %zd edges\n"
	 , inputFilename, nVertices, srcs.size());

  //initialize vertex data
  //convert to CSR to get the count of edges.
  std::vector<int> srcOffsets(nVertices + 1);
  std::vector<int> csrSrcs(srcs.size());
  edgeListToCSR<int>(
    nVertices, srcs.size(), &srcs[0], &dsts[0], &srcOffsets[0], 0, 0);

  std::vector<PageRank::VertexData> vertexData(nVertices);
  for( int i = 0; i < nVertices; ++i )
    vertexData[i].numOutEdges = srcOffsets[i + 1] - srcOffsets[i];

  std::vector<PageRank::VertexData> refVertexData;
  if( runTest )
  {
    printf("Running reference calculation\n");
    refVertexData = vertexData;
    run< GASEngineRef<PageRank> >(
      nVertices, &refVertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
    if( dumpResults )
    {
      printf("Reference\n");
      outputRanks(nVertices, &refVertexData[0]);
    }
  }

  run< GASEngineGPU<PageRank> >(
    nVertices, &vertexData[0], (int)srcs.size(), &srcs[0], &dsts[0]);
  if( dumpResults )
  {
    printf("GPU:\n");
    outputRanks(nVertices, &vertexData[0]);
  }

  if( runTest )
  {
    const float tol = 1.0e-6f;
    bool diff = false;
    for( int i = 0; i < nVertices; ++i )
    {
      if( fabs(vertexData[i].rank - refVertexData[i].rank) > tol )
      {
        printf("%d %f %f\n", i, refVertexData[i].rank, vertexData[i].rank);
        diff = true;
      }
    }
    if( diff )
      return 1;
    else
      printf("No differences found\n");
  }

  if( outputFilename )
  {
    FILE* f = fopen(outputFilename, "w");
    printf("writing results to file %s\n", outputFilename);
    outputRanks(nVertices, &vertexData[0], f);
    fclose(f);
  }

  free(inputFilename);
  free(outputFilename);

  return 0;
}

std::ostream& operator<<(std::ostream &out, const PageRank::VertexData &data) {
  out << data.rank;
  return out;
}
